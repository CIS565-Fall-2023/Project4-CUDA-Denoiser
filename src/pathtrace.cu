#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <chrono>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        //float timeToIntersect = gBuffer[index].t * 256.0;

        pbo[index].w = 0;
        //pbo[index].x = timeToIntersect;
        //pbo[index].y = timeToIntersect;
        //pbo[index].z = timeToIntersect;
        pbo[index].x = glm::abs(gBuffer[index].normal.x * 255.0);
        pbo[index].y = glm::abs(gBuffer[index].normal.y * 255.0);
        pbo[index].z = glm::abs(gBuffer[index].normal.z * 255.0);
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static glm::vec3* dev_inputImage = NULL;
static glm::vec3* dev_outputImage = NULL;
static glm::vec3* dev_diff = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static __constant__ float filter[5][5] = {
    {0.0039, 0.0156, 0.0234, 0.0156, 0.0039},
    {0.0156, 0.0625, 0.0938, 0.0625, 0.0156},
    {0.0234, 0.0938, 0.1406, 0.0938, 0.0234},
    {0.0156, 0.0625, 0.0938, 0.0625, 0.0156},
    {0.0039, 0.0156, 0.0234, 0.0156, 0.0039}
};

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    // TODO: initialize any extra device memeory you need
    hipMalloc(&dev_inputImage, pixelcount * sizeof(glm::vec3));
    hipMalloc(&dev_outputImage, pixelcount * sizeof(glm::vec3));
    //hipMalloc(&dev_diff, pixelcount * sizeof(glm::vec3));
    //hipMemset(dev_diff, 0, pixelcount * sizeof(glm::vec3));

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_gBuffer);
    // TODO: clean up any extra device memory you created
    hipFree(dev_inputImage);
    hipFree(dev_outputImage);
    //hipFree(dev_diff);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeSimpleMaterials (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];
    if (segment.remainingBounces == 0) {
      return;
    }

    if (intersection.t > 0.0f) { // if the intersection exists...
      segment.remainingBounces--;
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
        segment.remainingBounces = 0;
      }
      else {
        segment.color *= materialColor;
        glm::vec3 intersectPos = intersection.t * segment.ray.direction + segment.ray.origin;
        scatterRay(segment, intersectPos, intersection.surfaceNormal, material, rng);
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      segment.color = glm::vec3(0.0f);
      segment.remainingBounces = 0;
    }

    pathSegments[idx] = segment;
  }
}

__global__ void generateGBuffer (
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
  GBufferPixel* gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    //gBuffer[idx].t = shadeableIntersections[idx].t;
      gBuffer[idx].normal = shadeableIntersections[idx].surfaceNormal;
      gBuffer[idx].position = shadeableIntersections[idx].t * pathSegments[idx].ray.direction + pathSegments[idx].ray.origin;
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Pathtracing Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * NEW: For the first depth, generate geometry buffers (gbuffers)
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally:
    //     * if not denoising, add this iteration's results to the image
    //     * TODO: if denoising, run kernels that take both the raw pathtraced result and the gbuffer, and put the result in the "pbo" from opengl

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

    // Empty gbuffer
    hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    bool iterationComplete = false;
	while (!iterationComplete) {

	    // tracing
	    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	    computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
		    depth
		    , num_paths
		    , dev_paths
		    , dev_geoms
		    , hst_scene->geoms.size()
		    , dev_intersections
		    );
	    checkCUDAError("trace one bounce");
	    hipDeviceSynchronize();

        if (depth == 0) {
            generateGBuffer<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_intersections, dev_paths, dev_gBuffer);
        }

	    depth++;

        shadeSimpleMaterials<<<numblocksPathSegmentTracing, blockSize1d>>> (
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
            );
        iterationComplete = depth == traceDepth;
	}

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
    // Otherwise, screenshots are also acceptable.
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_gBuffer);
}

void showImage(uchar4* pbo, int iter) {
const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
}

void showImageAfterDenoising(uchar4* pbo, int iter) {
    const Camera& cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_inputImage);
}

__device__ float computeW(glm::vec3 p, glm::vec3 q, float sigma, int stepwidth) {
    return min(exp(-glm::dot((p - q), (p - q)) / (stepwidth * stepwidth) / (sigma * sigma)), 1.0f);
}

__global__ void applyFilter(glm::ivec2 resolution,
    glm::vec3 *dev_inputImage,
    glm::vec3* dev_outputImage,
    int stepwidth,
    float sigma_c,
    float sigma_n,
    float sigma_p,
    GBufferPixel *dev_gBuffer
    )
{
    // current index
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        float k = 0.0;
        int idx = x + resolution.x * y;
        glm::vec3 cum_sum = glm::vec3(0.f);
        // traverse all 25 pixels in the filter
        for (int i = -2; i < 3; i++) {
            for (int j = -2; j < 3; j++) {
                int q_x = glm::clamp(x + i * stepwidth, 0, resolution.x - 1);
                int q_y = glm::clamp(y + j * stepwidth, 0, resolution.y - 1);
                int q_idx = q_x + q_y * resolution.x;
                float w_rt = computeW(dev_inputImage[idx], dev_inputImage[q_idx], sigma_c, 1);
                float w_n = computeW(dev_gBuffer[idx].normal, dev_gBuffer[q_idx].normal, sigma_n, 1);
                float w_p = computeW(dev_gBuffer[idx].position, dev_gBuffer[q_idx].position, sigma_p, 1);
                float h_q = filter[i + 2][j + 2];
                float w_total = h_q * w_rt * w_n * w_p;
                
                k += w_total;
                cum_sum += w_total * dev_inputImage[q_idx];
            }
        }
        dev_outputImage[idx] = cum_sum / k;
    }
}

//__global__ void storeDiff(glm::ivec2 resolution, glm::vec3 *dev_inputImage, glm::vec3* dev_outputImage, glm::vec3* dev_diff) {
//    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
//    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//
//    if (x < resolution.x && y < resolution.y) {
//        int idx = x + resolution.x * y;
//        dev_diff[idx] += dev_inputImage[idx] - dev_outputImage[idx];
//    }
//}
//
//__global__ void reconstruction(glm::ivec2 resolution, glm::vec3* dev_inputImage, glm::vec3* dev_diff) {
//    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
//    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//
//    if (x < resolution.x && y < resolution.y) {
//        int idx = x + resolution.x * y;
//        dev_inputImage[idx] += dev_diff[idx];
//    }
//}

void denoise(int ui_filterSize,
    float ui_colorWeight,
    float ui_normalWeight,
    float ui_positionWeight) 
{
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // copy the original image
    hipMemcpy(dev_inputImage, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

    // calculate number of levels based on filter size
    int n = glm::floor(std::log2f((ui_filterSize - 1) * 0.25)) + 1;
    //cout << "number of level: " << n << endl;

    // timer start
    const auto start = std::chrono::high_resolution_clock::now();
    // loop to apply a tous filter on image
    for (int i = 0; i < n; i++) {
        //cout << "\ti: " << i << endl;
        int stepwidth = 1 << i;
        applyFilter <<<blocksPerGrid2d, blockSize2d>>> (cam.resolution,
            dev_inputImage,
            dev_outputImage,
            stepwidth,
            ui_colorWeight,
            ui_normalWeight,
            ui_positionWeight,
            dev_gBuffer
            );;
        

        //storeDiff<<<blocksPerGrid2d, blockSize2d >>> (cam.resolution, dev_inputImage, dev_outputImage, dev_diff);
        
        // swap the buffer
        std::swap(dev_inputImage, dev_outputImage);
    }
    hipDeviceSynchronize();
    // timer end
    const auto end = std::chrono::high_resolution_clock::now();
    auto diff = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    cout << "duration: " << diff << endl;

    hipMemcpy(hst_scene->state.image.data(), dev_inputImage, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    //reconstruction <<<blocksPerGrid2d, blockSize2d >>> (cam.resolution, dev_inputImage, dev_diff);
}