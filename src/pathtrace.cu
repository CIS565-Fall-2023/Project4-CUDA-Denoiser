#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h" 
#include "interactions.h"
#include "bvh.h"
#include "cuda_timer.h"

#define ERRORCHECK 1
#define DEBUG_NORM 0
#define DEBUG_POS 0
#define DEBUG_BLUR 0

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}


struct is_done
{
	__host__ __device__
		bool operator()(const PathSegment& seg)
	{
		return seg.remainingBounces == 0;
	}
};


__global__ void updateMaterialKey(int num_paths, ShadeableIntersection* in_intersects, int* out_materialKey) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < num_paths) {
		out_materialKey[idx] = in_intersects[idx].materialId;
	}
}


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, float iter, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] = glm::mix(image[iterationPath.pixelIndex],iterationPath.color,1.f/iter);
	}
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, glm::vec3* image, float iter) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		
		//glm::vec3 pix = image[index];
		glm::vec3 pix = glm::pow(image[index]/(image[index] + 1.f),glm::vec3(1.f/2.2));
	

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z * 255.0), 0, 255);
		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

__global__ void transformTriangles(int num_trigs, Mesh* in_meshs, Triangle* out_trigs) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= num_trigs)return;
	Triangle& trig = out_trigs[idx];
	Mesh& mesh = in_meshs[trig.meshId];
	
	trig.v1.pos = glm::vec3(mesh.transform * glm::vec4(trig.v1.pos,1));
	trig.v1.normal = glm::normalize(glm::mat3(mesh.invTranspose) * trig.v1.normal);

	trig.v2.pos = glm::vec3(mesh.transform * glm::vec4(trig.v2.pos, 1));
	trig.v2.normal = glm::normalize(glm::mat3(mesh.invTranspose) * trig.v2.normal);

	trig.v3.pos = glm::vec3(mesh.transform * glm::vec4(trig.v3.pos, 1));
	trig.v3.normal = glm::normalize(glm::mat3(mesh.invTranspose) * trig.v3.normal);
}

void PathTracer::initMeshTransform()
{
	//cout << "Transform triangles based on transformation matrix" << endl;
	int blockSize1d = 128;
	int N = dev_trigs.size();
	dim3 numblocksTransformTrigs = (N + blockSize1d - 1) / blockSize1d;
	transformTriangles<<<numblocksTransformTrigs,blockSize1d>>>(N, dev_geoms.get(), dev_trigs.get());
}
/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void PathTracer::initDataContainer(GuiDataContainer* guiData)
{
	m_guiData = guiData;
}
void PathTracer::pathtraceInit(Scene* scene)
{
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;
	
	this->dev_img.malloc(pixelcount, "Malloc dev_img error");
	hipMemset(this->dev_img.get(), 0, pixelcount * sizeof(glm::vec3));
	
	this->dev_path.malloc(pixelcount, "Malloc dev_path error");

	this->dev_mat.malloc(scene->materials.size(), "Malloc material error");
	hipMemcpy(this->dev_mat.get(), scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	this->dev_intersect.malloc(pixelcount, "Malloc dev_intersect error");

	this->dev_geoms.malloc(scene->meshs.size(), "Malloc dev_mesh error");
	hipMemcpy(this->dev_geoms.get(), scene->meshs.data(), scene->meshs.size() * sizeof(Mesh), hipMemcpyHostToDevice);

	this->dev_trigs.malloc(scene->trigs.size(), "Malloc dev_trigs error");
	hipMemcpy(this->dev_trigs.get(), scene->trigs.data(), scene->trigs.size() * sizeof(Triangle), hipMemcpyHostToDevice);
	initMeshTransform();

	this->dev_texObjs.malloc(scene->texs.size(), "Malloc dev_texObjs error");
	std::vector<hipTextureObject_t> texObjs;
	for (auto& texObj : scene->texs) {
		texObjs.push_back(texObj.m_texObj);
	}
	hipMemcpy(this->dev_texObjs.get(), texObjs.data(), texObjs.size() * sizeof(hipTextureObject_t), hipMemcpyHostToDevice);

	if (scene->envTexId != -1) {
		hasEnvMap = true;
		envMap = scene->texs[scene->envTexId].m_texObj;
	}
	//get transformed triangles to CPU to build BVH tree, triangles will be reordered based on tree
	std::vector<Triangle> tmp_trig(scene->trigs.size());
	hipMemcpy(tmp_trig.data(), this->dev_trigs.get(), scene->trigs.size() * sizeof(Triangle), hipMemcpyDeviceToHost);
	BVHTreeBuilder builder;
	auto bvh = builder.buildBVHTree(tmp_trig);
	//sent ordered triangle back to GPU
	hipMemcpy(this->dev_trigs.get(), tmp_trig.data(), scene->trigs.size() * sizeof(Triangle), hipMemcpyHostToDevice);
	this->dev_bvh.malloc(bvh.size(), "Malloc dev_bvh error");
	hipMemcpy(this->dev_bvh.get(), bvh.data(), bvh.size() * sizeof(BVHNode), hipMemcpyHostToDevice);
	this->dev_donePaths.malloc(pixelcount, "Malloc dev_donePath error");

	dev_gbuffer.malloc(pixelcount, "Malloc dev_gbuffer error");
	hipMemset(dev_gbuffer.get(), 0, pixelcount * sizeof(GBuffer));
	dev_gImg0.malloc(pixelcount, "Malloc dev_gImg error");
	hipMemset(dev_gImg0.get(), 0, pixelcount * sizeof(glm::vec3));
	dev_gImg1.malloc(pixelcount, "Malloc dev_gImg error");
	hipMemset(dev_gImg1.get(), 0, pixelcount * sizeof(glm::vec3));

	checkCUDAError("pathtraceInit");
}


// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* in_paths
	, Triangle* in_trigs
	, Mesh* in_meshs
	, int num_trigs
	, BVHNode* in_bvh
	, int num_bvh
	, ShadeableIntersection* out_intersects
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment seg = in_paths[path_index];
		
		float t;
		float t_min = 100000.f;
		
		glm::vec3 bary; // for interpolation
		glm::vec3 tmp_bary;
		int hit_trig_index = -1;
		int needToVisit[32];
		int stackTop = 0;
		needToVisit[0] = 0;

		while (stackTop >= 0 && stackTop < 32) {
			int idxToVis = needToVisit[stackTop--];
			BVHNode node = in_bvh[idxToVis];
			bool hit = AABBIntersectionTest(seg.ray.origin, seg.ray.direction, node.boundingBox,t);
			if (hit && t_min > t) 
			{ 
				if (node.primNum == 0) {
					//second child
					int firstChild = idxToVis + 1;
					int secondChild = -1;
					if (node.secondChildOffset != -1) {
						secondChild = idxToVis + node.secondChildOffset;
						if (seg.ray.direction[node.axis] > 0) {
							needToVisit[++stackTop] = secondChild;
							needToVisit[++stackTop] = firstChild;
						}
						else {
							needToVisit[++stackTop] = firstChild;
							needToVisit[++stackTop] = secondChild;
						}
					}
					else {
						needToVisit[++stackTop] = firstChild;
					}
					
				}
				else { //every thing in bounding box will have larger t
					for (int i = 0;i < node.primNum;++i) {
						const Triangle& trig = in_trigs[node.firstPrimId + i];
						hit = triangleIntersectionTest(seg.ray.origin, seg.ray.direction, trig.v1.pos, trig.v2.pos, trig.v3.pos, tmp_bary,t);
						if (hit && t_min > t)
						{
							t_min = t;
							bary = tmp_bary;
							hit_trig_index = node.firstPrimId + i;
						}
					}
				}
			}
		}

		if (hit_trig_index == -1)
		{
			out_intersects[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			const Triangle& trig = in_trigs[hit_trig_index];
			out_intersects[path_index].t = t_min;
			out_intersects[path_index].materialId = in_meshs[trig.meshId].materialId;
			out_intersects[path_index].surfaceNormal = glm::normalize(trig.v1.normal * bary.x + trig.v2.normal * bary.y + trig.v3.normal * bary.z);
			out_intersects[path_index].surfaceUV = trig.v1.uv * bary.x + trig.v2.uv * bary.y + trig.v3.uv * bary.z;
		}
	}
}


__global__ void processPBR(
	int iter, int depth
	, int num_paths
	, ShadeableIntersection* intersections
	, PathSegment* pathSegments
	, Material* materials
	, hipTextureObject_t* textures
	, hipTextureObject_t envMapTex
	, bool hasEnvMap
)
{
	int path_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_idx >= num_paths)return; // no light
	PathSegment& seg = pathSegments[path_idx];
	ShadeableIntersection& intersect = intersections[path_idx];
	if (seg.remainingBounces < 1) // end bounce
	{
		return;
	}
	--seg.remainingBounces;
	if (intersect.t <= 0.) // no intersection
	{
		if (hasEnvMap) {
			seg.color *= getEnvLight(seg.ray.direction, envMapTex);
		}
		else {
			seg.color = glm::vec3(0.f);
		}
		seg.remainingBounces = 0;
		return;
	}
	Material material = materials[intersect.materialId];
	glm::vec3 normal = intersect.surfaceNormal;
	if (material.bumpId!=-1) {
		normal = tangentToWorld(normal) * texture2D(intersect.surfaceUV, textures[material.bumpId]);
	}

	if (material.emittance > 0) // hit light
	{
		seg.color *= (material.color * material.emittance);
		seg.remainingBounces = 0;
		return;
	}
	
	if (seg.remainingBounces < 1) // end bounce and didn't hit light
	{
		seg.color = glm::vec3(0.);
		return;
	}

	float pdf = 1.f;

	//treat it like normal ray intersection condition
	seg.ray.origin = intersect.t * seg.ray.direction + seg.ray.origin;
	

	glm::vec3 wo = -seg.ray.direction;

	thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_idx, depth);


	if (!sampleRay(wo, intersect.surfaceNormal, material, rng, pdf, seg.ray.direction,seg)) {
		//this is a ray need to be discarded
		seg.remainingBounces = 0;
		seg.color = glm::vec3(0.);
		return;
	}

	//fix strange artifact
	seg.ray.origin += 0.01f * seg.ray.direction;

	glm::vec3 bsdf = getBSDF(seg.ray.direction, wo, intersect.surfaceUV, material,textures);
	//           albedo           absdot
	seg.color *= (bsdf * glm::clamp(abs(glm::dot(normal, seg.ray.direction)), 0.f, 1.f) / pdf);
	
}


//store value that can be get in first frame
__global__ void processGBuffer(
	float iter
	, int num_paths
	, PathSegment* in_segments 
	, ShadeableIntersection* in_intersects
	, Material* in_materials
	, hipTextureObject_t* in_textures
	, GBuffer* out_gbuffer
) {
	int path_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_idx >= num_paths)return;
	PathSegment& seg = in_segments[path_idx];
	ShadeableIntersection& intersect = in_intersects[path_idx];
	if (intersect.t <= 0.) // no intersection
	{
		return;
	}
	Material material = in_materials[intersect.materialId];
	glm::vec3 normal = intersect.surfaceNormal;
	if (material.bumpId != -1) {
		normal = tangentToWorld(normal) * texture2D(intersect.surfaceUV, in_textures[material.bumpId]);
	}
	GBuffer& gbuffer = out_gbuffer[seg.pixelIndex];
	gbuffer.norm = glm::mix(gbuffer.norm,normal,1.f/iter);
	gbuffer.pos = glm::mix(gbuffer.pos ,intersect.t * seg.ray.direction + seg.ray.origin, 1.f/iter);
}

//make denosiser
inline __device__ bool inScreen(
	int x
	, int y
	, int resX
	, int resY
) {
	return x >=0 && x<resX&& y>=0 && y < resY;
}
__device__ int to1D(
	int x
	, int y
	, int resX
	, int resY
) {
	return resX * y + x;
}
__global__ void denoise(
	int resX
	, int resY
	, GBuffer* in_gbuffer
	, int offset
	, float c_phi
	, float n_phi
	, float p_phi
	, glm::vec3* in_c
	, glm::vec3* out_c
) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (!inScreen(x, y, resX, resY))return;
	int idx = to1D(x, y, resX, resY);
	float hs[5] = { 0.0625,0.25,0.375,0.25,0.0625 };
	glm::vec3 sum(0.f);
	float weightSum = 0.f;

	GBuffer& gval = in_gbuffer[idx];
	glm::vec3 cval = in_c[idx];

#if DEBUG_BLUR
	for (int i = -2;i <= 2;++i) {
		for (int j = -2;j <= 2;++j) {
			int tmpX = x + i * offset;
			int tmpY = y + j * offset;
			if (inScreen(tmpX, tmpY, resX, resY)) {
				int tmpIdx = to1D(tmpX, tmpY, resX, resY);
				glm::vec3 tmpC = in_c[tmpIdx];
				float weight = 1.f;

				float kernel = hs[i + 2] * hs[j + 2];
				sum += tmpC * weight * kernel;
				weightSum += weight * kernel;
			}
		}
	}
	out_c[idx] = sum / weightSum;
	return;
#endif

#if DEBUG_NORM
	out_c[idx] = gval.norm * 0.5f + glm::vec3(0.5f);
	return;
#endif

#if DEBUG_POS
	out_c[idx] = gval.pos * 0.2f;
	return;
#endif

	for (int i = -2;i <= 2;++i) {
		for (int j = -2;j <= 2;++j) {
			int tmpX = x + i * offset;
			int tmpY = y + j * offset;
			if (inScreen(tmpX, tmpY, resX, resY)) {
				int tmpIdx = to1D(tmpX, tmpY, resX, resY);
				glm::vec3 tmpC = in_c[tmpIdx];
				GBuffer tmpG = in_gbuffer[tmpIdx];

				glm::vec3 tmpVal = cval - tmpC;
				float dist2 = glm::dot(tmpVal,tmpVal);
				float c_w = min(exp(-(dist2) / c_phi), 1.f);
				
				tmpVal = gval.norm - tmpG.norm;
				dist2 = max(glm::dot(tmpVal, tmpVal) / float( offset * offset), 0.f);
				float n_w = min(exp(-(dist2) / n_phi), 1.f);

				tmpVal = gval.pos - tmpG.pos;
				dist2 = glm::dot(tmpVal, tmpVal);
				float p_w = min(exp(-(dist2) / p_phi), 1.f);
				float weight = c_w * n_w * p_w;
				
				float kernel = hs[i + 2] * hs[j + 2];
				sum += tmpC * weight * kernel;
				weightSum += weight * kernel;
			}
		}
	}
	out_c[idx] = sum / weightSum;
}


void PathTracer::pathtrace(uchar4* pbo, int frame, int iter)
{
	const int traceDepth = hst_scene->state.traceDepth;

	const Camera& cam = hst_scene->state.camera;

	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);
	

	// 1D block for path tracing
	const int blockSize1d = 128;

	PathSegment* dev_paths = this->dev_path.get();
	PathSegment* dev_donePaths = this->dev_donePaths.get();
	int* dev_materialId = this->dev_materialId.get();
	ShadeableIntersection* dev_intersections = this->dev_intersect.get();
	Mesh* dev_meshs = this->dev_geoms.get();
	Triangle* dev_trigs = this->dev_trigs.get();
	int num_trigs = hst_scene->trigs.size();
	Material* dev_materials = this->dev_mat.get();
	GuiDataContainer* guiData = this->m_guiData;
	glm::vec3* dev_image = this->dev_img.get();
	BVHNode* dev_bvh = this->dev_bvh.get();
	int num_bvh = this->dev_bvh.size();
	hipTextureObject_t* dev_texObjs = this->dev_texObjs.get();
	float lenRadius = m_guiData->lensRadius;
	float focusLen = m_guiData->focusLength;

	thrust::device_ptr<PathSegment> thrust_paths(dev_paths);
	thrust::device_ptr<PathSegment> thrust_paths_end(dev_paths + pixelcount);
	thrust::device_ptr<PathSegment> thrust_donePaths(dev_donePaths);
	thrust::device_ptr<int> thrust_materialId(dev_materialId);
	int num_paths = thrust_paths_end - thrust_paths;
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	int depth = 0;
	bool iterationComplete = false;
	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth,lenRadius, focusLen, dev_paths);
	checkCUDAError("generate camera ray");
	while (!iterationComplete) {

		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_paths
			, dev_paths
			, dev_trigs
			, dev_meshs
			, num_trigs
			, dev_bvh
			, num_bvh
			, dev_intersections
			);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		if (m_guiData->ui_denoise) {
			if (depth == 0) {
				processGBuffer << <numblocksPathSegmentTracing, blockSize1d >> > (
					iter
					, pixelcount
					, dev_paths
					, dev_intersections
					, dev_materials
					, dev_texObjs
					, dev_gbuffer.get()
					);
				checkCUDAError("trace gbuffer");
				hipDeviceSynchronize();
			}
		}
		processPBR << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter, depth
			, num_paths
			, dev_intersections
			, dev_paths
			, dev_materials
			, dev_texObjs
			, envMap 
			, hasEnvMap
			);
		depth++;

		// * TODO: Stream compact away all of the terminated paths.
		thrust_donePaths = thrust::copy_if(thrust_paths, thrust_paths_end, thrust_donePaths, is_done());
		thrust_paths_end = thrust::remove_if(thrust_paths, thrust_paths_end, is_done());
		num_paths = thrust_paths_end - thrust_paths;
		iterationComplete = (num_paths  == 0);

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	
	finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, iter, dev_image, dev_donePaths);
	if (m_guiData->ui_denoise) {
		glm::vec3* o_dColor = dev_gImg0.get();
		glm::vec3* i_dColor = dev_gImg1.get();
		GBuffer* i_gbuffer = dev_gbuffer.get();
		//gatherGBufferColor << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_donePaths, o_dColor);
		hipMemcpy(o_dColor, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
		for (int i = 0;i < m_guiData->ui_filterSize;++i) {
			int offset = 1 << i;
			swap(o_dColor, i_dColor);
			denoise << <blocksPerGrid2d, blockSize2d >> > (cam.resolution.x, cam.resolution.y, i_gbuffer, offset, m_guiData->ui_colorWeight, m_guiData->ui_normalWeight, m_guiData->ui_positionWeight, i_dColor, o_dColor);
		}
		// Send results to OpenGL buffer for rendering
		sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, o_dColor, iter);

		// Retrieve image from GPU
		hipMemcpy(hst_scene->state.image.data(), o_dColor,
			pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	}
	else {
		// Send results to OpenGL buffer for rendering
		sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, dev_image, iter);

		// Retrieve image from GPU
		hipMemcpy(hst_scene->state.image.data(), dev_image,
			pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	}
	///////////////////////////////////////////////////////////////////////////


	checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_gBuffer);
}

void showImage(uchar4* pbo, int iter) {
const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
}
